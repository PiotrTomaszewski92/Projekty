#include "hip/hip_runtime.h"
#include <iostream>
#include <stdio.h>
#include <omp.h>
#include <opencv/cv.h>
#include <opencv/highgui.h>

using namespace cv;
using namespace std;

//-----------------------------------------------------------------------------------------
// zdefiniowanie tablicy o statycznym rozmiarze 5x5
__constant__ int maskaGPU[5][5];


//-----------------------------------------------------------------------------------------
// funkcja rozmywajaca zadane zdjecie
// __global__ - kwalifikator - informuje kompilator, ze dana funkcja powinna byc skompilowana dla urzadzenia a nie dla hosta

__global__ void rozmycie(int kanal_zdj, int sumaWag, unsigned char* zdjWe, unsigned char* zdjWy, int wysokosc, int rozmiar) 
{
	int id = blockIdx.x * blockDim.x + threadIdx.x;
	double piksel= 0;
	int indeks,x,y;

			for(x = 0; x < 5; x++ ) 
			{
				for(y = 0; y < 5; y++) 
				{
					indeks = id + ((y - 2) * rozmiar) + ((x - 2) * kanal_zdj);
					piksel+= maskaGPU[x][y] * zdjWe[indeks];
				}
			}
			zdjWy[id] = (unsigned char)(piksel/sumaWag);
			id += blockDim.x * gridDim.x;
			piksel = 0;
}

//-----------------------------------------------------------------------------------------
// funkcja sumujaca maski
int sumaMaski(int maska[5][5], int x, int y, int suma){
	for (x=0; x<5;x++){
		for (y=0; y<5;y++){
			suma+=maska[x][y];
		}
	}
	return suma;
}

int main(int argc, char** argv )
{
/*-------------------------------------------------------------------------------------
1. Definiowanie zmiennych.
-------------------------------------------------------------------------------------*/	
	// zmienne do wyznaczenia czasu wykonania zadania (deklaracja zdarzen)
	hipEvent_t czas_start, czas_stop;
	hipEventCreate(&czas_start);
	hipEventCreate(&czas_stop);
	
	unsigned char *zdj_gpu_we, *zdj_gpu_wy;

/*-------------------------------------------------------------------------------------
2. Deklaracja maski w postaci 2-wymiarowej tablicy liczb całkowitych.
-------------------------------------------------------------------------------------*/		
  int maska[5][5] ={
  	{1,1,2,1,1},
  	{1,2,4,2,1},
  	{2,4,8,4,2},
  	{1,2,4,2,1},
  	{1,1,2,1,1}
  };
  
/*-------------------------------------------------------------------------------------
3. Kopiowanie pamięci tablicy maskaGPU do $maska. Kopiowanie danych do pamięci stałej.

	hipMemcpyToSymbol()
		maskaGPU 		- symbol (zmienna) docelowa na urządzeniu
		$maska 	 		- adres źródłowy pamieci
		sizeof(int)*5*5 - rozmiar pamieci w bajtach
		
-------------------------------------------------------------------------------------*/  
	hipMemcpyToSymbol(HIP_SYMBOL(maskaGPU), &maska, sizeof(int) * 5 * 5);
	
	
/*-------------------------------------------------------------------------------------
4. Warunek sprawdzajacy czy istnieja wszystkie niezbedne argumenty wejsciowe.
-------------------------------------------------------------------------------------*/
	if ( argc != 3 )
	{
		cout << "0 ms - brak wszystkich argumentow" << endl;
		return 0;
	}

/*-------------------------------------------------------------------------------------
5. Wczytanie pliku do zmiennej "zdj_we".
-------------------------------------------------------------------------------------*/  
 
	Mat zdj_we,zdj_wy;
  
	zdj_we = imread(argv[1], CV_LOAD_IMAGE_COLOR);
  
	if ( !zdj_we.data )
	{
      cout <<  "0 ms - problem z ladowaniem wartosci" << endl ;
      return 0;
	}
	
/*-------------------------------------------------------------------------------------
6. Podział zadania na gridy oraz bloki.
-------------------------------------------------------------------------------------*/  
 	
	int rozmiarBlok = 32;
	int rozmiarSiatkaSzer = zdj_we.cols/rozmiarBlok;
	int rozmiarSiatkaDlug = zdj_we.rows/rozmiarBlok;

	
	dim3 siatka(rozmiarSiatkaSzer, rozmiarSiatkaDlug);
	dim3 blok(rozmiarBlok, rozmiarBlok);
	
/*-------------------------------------------------------------------------------------
7. Utworzenie kopii wcześniej wczytanego zdjęcia do zmiennej zdj_wy.
-------------------------------------------------------------------------------------*/	
	
	zdj_wy = zdj_we.clone();

/*-------------------------------------------------------------------------------------
8. Alokowanie pamieci na karcie graficznej.

	hipMalloc 
		(void**) - wskaznik wskazuje adres nowo alokowanej pamieci
		sizeof() - rozmiar alokowanej pamieci
-------------------------------------------------------------------------------------*/	  

	hipMalloc(&zdj_gpu_we, zdj_we.rows*zdj_we.step*sizeof(unsigned char));
	hipMalloc(&zdj_gpu_wy, zdj_we.rows*zdj_we.step*sizeof(unsigned char));

/*-------------------------------------------------------------------------------------
9. Kopiowanie danych miedzy pamieci RAM do karty graficznej.

	hipMemcpy() 
		zdj_gpu_we 				 - wskaźnik na obszar pamięci, do której nastąpi kopiowanie
		zdj_we.ptr()			 - wskaźnik na obszar pamięci, z której nastąpi kopiowanie
		zdj_we.rows*zdj_we.step  - liczba bajtów do skopiowania
		hipMemcpyHostToDevice   - obszar pamięci źródłowej należy do komputera (RAM), natomiast docelowy obszar pamięci należy do pamięci karty graficznej
	
-------------------------------------------------------------------------------------*/		

	hipMemcpy(zdj_gpu_we, zdj_we.ptr(), zdj_we.rows*zdj_we.step, hipMemcpyHostToDevice);

/*-------------------------------------------------------------------------------------
10. Rozpoczecie pomiaru czasu obliczen.
-------------------------------------------------------------------------------------*/		

	hipEventRecord(czas_start, 0);

	rozmycie<<<siatka,blok>>>(zdj_we.channels(), sumaMaski(maska,0,0,0), zdj_gpu_we, zdj_gpu_wy, zdj_we.rows, zdj_we.step);

	hipEventRecord(czas_stop, 0);
	hipEventSynchronize(czas_stop);
	float czas = 0.0;
	hipEventElapsedTime(&czas, czas_start, czas_stop);

	// blokuje bieżący wątek aplikacji do czasu zakończenia wszystkich oczekiwanych obliczeń na karcie graficznej.
	hipDeviceSynchronize();

/*-------------------------------------------------------------------------------------
11. Kopiowanie danych z karty graficznej do pamieci RAM.

	hipMemcpy() 
		zdj_wy.ptr() 			 - wskaźnik na obszar pamięci, do której nastąpi kopiowanie
		zdj_gpu_wy			 	 - wskaźnik na obszar pamięci, z której nastąpi kopiowanie
		zdj_we.rows*zdj_we.step  - liczba bajtów do skopiowania
		hipMemcpyDeviceToHost   - obszar pamięci źródłowej należy do pamięci karty graficznej, natomiast docelowy obszar pamięci należy do komputera (RAM)
	
-------------------------------------------------------------------------------------*/		
	hipMemcpy(zdj_wy.ptr(), zdj_gpu_wy, zdj_we.rows*zdj_we.step, hipMemcpyDeviceToHost);

	
/*-------------------------------------------------------------------------------------
12. Zwolnienie wcześniej zaalokowanej pamięci na karcie graficznej.
-------------------------------------------------------------------------------------*/			
	hipFree(zdj_gpu_wy);
	hipFree(zdj_gpu_we);

/*-------------------------------------------------------------------------------------
13. Zapisanie zmodyfikowanego obrazka w miejsce zadane jako drugi argument.
-------------------------------------------------------------------------------------*/
	imwrite(argv[2], zdj_wy);
	printf("Czas: %.3f ms", czas);
	
   return 0;
}
